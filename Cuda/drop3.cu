#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h> 
#include "float.h"
#include <float.h>
#include <thrust/sort.h>
#include "HeapData.h"

#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

extern "C" {

	__global__ void findNeighbours(
		float * vectors,
		int vectorCount,
		int atributeCount,
		int neighborCount,
		int* classes,
		HeapData* heapMemory,
		float* nearestEnemyDistance
	) {

		const int id = blockIdx.x*blockDim.x + threadIdx.x;
		if (id >= vectorCount) return;

		nearestEnemyDistance[id] = FLT_MAX;

		HeapData* heap = heapMemory + (id*neighborCount);

		const float* currentRow = vectors + (atributeCount*id);

		for (int i = 0; i < neighborCount; i++) {
			heap[i].val = FLT_MAX;
		}

		for (int i = 0; i < vectorCount; i++)
		{
			const float* row = vectors + (atributeCount*i);
			if (row != currentRow) {
				float result = 0.f;
				for (int j = 0; j < atributeCount; j++)
				{
					const float d = currentRow[j] - row[j];
					result += d*d;
				}
				result = sqrtf(result);

				if (classes[i] != classes[id]    //is eneamy
					&& result < nearestEnemyDistance[id]
					)
				{
					nearestEnemyDistance[id] = result;
				}

				if (result < heap[0].val) {
					heap[0].val = result;
					heap[0].label = i;
					hipify(heap, neighborCount);
				}
			}
		}

		thrust::sort(heap, heap + neighborCount, HeapCompare());

	}





	__global__ void calculateDistances(
		float * vectors,
		int vectorCount,
		int vectorToExamine,
		int atributeCount,
		unsigned char* vectorsInDataset,
		float* results
	) {

		const int id = blockIdx.x*blockDim.x + threadIdx.x;
		if (id >= vectorCount) return;
		if (vectorsInDataset[id] == 0) return;

		const float* examinedVector = vectors + (atributeCount*vectorToExamine);
		const float* other = vectors + (atributeCount*id);

		float result = 0.f;
		for (int j = 0; j < atributeCount; j++)
		{
			const float d = examinedVector[j] - other[j];
			result += d*d;
		}
		result = sqrtf(result);
		results[id] = result;

	}

}