

#include <hip/hip_runtime.h>
 
#include <builtin_types.h> 

#include <float.h>




#include <math.h>


extern "C" {

	__constant__ int genLength;


	__global__ void countVectors(
		unsigned char* gens,
		int* vectorSizes
	) {

		const int id = threadIdx.x + blockIdx.x * blockDim.x;
		//if (id >= popSize) return;

		vectorSizes[id] = 0;
		const unsigned char* currentGen = gens + genLength*id;

		for (int i = 0; i < genLength; i++)
		{
			if (currentGen[i])
				vectorSizes[id]++;
		}

	}



}